#include "hip/hip_runtime.h"
/*
 * Path Traced Virtual Textures (PTVT)
 * Copyright 2018 Hans Cronau
 *
 * File based on the Optix SDK optixPathTracer sample,
 * Copyright 2016 Nvidia Corporation
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optixu/optixu_math_namespace.h>
#include "optixPathTracer.h"
#include "random.h"
#include "helpers.h"
#include <stdio.h>
#include "vtHelpers.h"
#include <sampleConfig.h>

using namespace optix;

struct PerRayData_pathtrace
{
    float3 result;
    float3 radiance;
    float3 attenuation;
    float3 origin;
    float3 direction;
    unsigned int seed;
    int depth;
    int countEmitted;
    int done;

    // Per ray support for virtual textures / path differentials:
    int within_fragment_path_id;     // required for virtual textures and path differentials
    float total_path_distance;       // required for virtual textures
    unsigned int differential_count; // required for differentials
};

struct PerRayData_pathtrace_shadow
{
    bool inShadow;
};

// Scene wide variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );

// Path differentials
rtDeclareVariable(uint, differentialDepth, , );
rtBuffer<float3, 3>     positionDifferentials;
rtBuffer<float3, 3>     directionDifferentials;

// Custom render modes
rtDeclareVariable(uint, limit_to_one_bounce, , );
rtDeclareVariable(uint, debug_render_mode, , );

//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float3, bad_color, , );
rtDeclareVariable(unsigned int, frame_number, , );
rtDeclareVariable(unsigned int, sqrt_num_samples, , );
rtDeclareVariable(unsigned int, rr_begin_depth, , );
rtDeclareVariable(unsigned int, pathtrace_ray_type, , );
rtDeclareVariable(unsigned int, pathtrace_shadow_ray_type, , );

rtBuffer<float4, 2>              output_buffer;
rtBuffer<ParallelogramLight>     lights;

RT_PROGRAM void pathtrace_camera()
{
    size_t2 screen = output_buffer.size();

    float2 inv_screen = 1.0f / make_float2(screen);
    float2 inv_screen_times_two = 2.0f * inv_screen;
    float2 pixel_screen = (make_float2(launch_index)) * inv_screen_times_two - 1.0f;
    // In the line above a pixel coordinate is created in screen space.
    // The launch_index is of type N^2 (incl. 0).
    // Is is converted to a screenspace in R^2 with the domain [-1.0, 1.0]^2.
    // Reason for the domain centered around 0.0 is that the camera's W vector points to the centre of the in-world screen (instead of for example to the top left).
    // The camera's U and V vectors in turn only span half the screen, from the centre to the edges.
    // pixel_screen.x and pixel_screen.y correspond respectively to x and y in Suykens2001 A.1 Pixel Sampling.

    float2 pixel_to_screen = inv_screen_times_two / sqrt_num_samples;
    // Scales pixel space to world space.
    // It is calculated here once so is doesn't have to be repeated in the loop below.
    // = (1.0 / sqrt_num_samples) * inv_screen_times_two

    unsigned int current_within_pixel_sample = sqrt_num_samples*sqrt_num_samples;
    // Counter for within-pixel samples.

    float3 result = make_float3(0.0f);

    unsigned int seed = tea<16>(screen.x*launch_index.y + launch_index.x, frame_number);
    do
    {
        //
        // Sample pixel using jittering
        //
        // Note: Samples are distributed evenly over pixel space. This, effectively, is supersampling.
        unsigned int current_within_pixel_sample_x = current_within_pixel_sample % sqrt_num_samples;
        unsigned int current_within_pixel_sample_y = current_within_pixel_sample / sqrt_num_samples;
        float2 current_within_pixel_sample_jittered = make_float2(
            current_within_pixel_sample_x - rnd(seed),
            current_within_pixel_sample_y - rnd(seed)
            );

        float2 point_in_screen = pixel_screen + current_within_pixel_sample_jittered * pixel_to_screen; // x * pix_w + pix_l (Suykens2001 A.1 Pixel Sampling)
        // In the line above I add a pixel's screen coordinate to the current within-pixel sample's coordinate, plus some random jitter.
        // The random values are within the range [0.0, 1.0] and subtracted from the absolute within-pixel sample's coordinates (domain [0, sqrt_num_samples)^2).
        // Combined they are scaled from (within-)pixel space to screen space (domain [-1, 1]^2) and added to the current pixel's screen space coordinate.
        // Notes on Igehy1999 and Suykens2001:
        // - The within-pixel coordinates in domain [0, 1]^2 are the unit random values x and y in Suykens2001.
        // - point_in_screen.x and point_in_screen.y are respectively called u and v in View + u*Right + v*Up.

        float3 ray_origin = eye;
        float3 ray_direction = normalize(point_in_screen.x*U + point_in_screen.y*V + W); // See Igehy1999 formula (4)

        // Initialze per-ray data
        PerRayData_pathtrace prd;
        prd.result = make_float3(0.f);
        prd.attenuation = make_float3(1.f);
        prd.countEmitted = true;
        prd.done = false;
        prd.seed = seed;
        prd.depth = 0;

        // Initialise VT data
        prd.within_fragment_path_id = current_within_pixel_sample - 1;
        prd.total_path_distance = 0.0f;

        // Initialise path differentials: create initial differentials
        prd.differential_count = 0;
        if (differentialDepth != 0) // TODO(HansCronau): Optimisation - We could assume differentialDepth is never < 1 to remove this if-statement.
        {
            const float3 dPdx = make_float3(0);
            const float3 dPdy = make_float3(0);
            // See Igehy1999 formula (8) and Suykens2001 A.1 Pixel Sampling
            const float3 dDdx = differential_generation_direction(ray_direction, U) * pixel_to_screen.x; // = dDdu * pix_w (Suykens2001 A.1 Pixel Sampling)
            const float3 dDdy = differential_generation_direction(ray_direction, V) * pixel_to_screen.y; // = dDdv * pix_h (Suykens2001 A.1 Pixel Sampling)

            // Determine index within differential buffers.
            const uint3 bufferIndexX = make_uint3(launch_index, prd.differential_count + 0);
            const uint3 bufferIndexY = make_uint3(launch_index, prd.differential_count + 1);
            positionDifferentials[bufferIndexX] = dPdx;
            positionDifferentials[bufferIndexY] = dPdy;
            directionDifferentials[bufferIndexX] = dDdx;
            directionDifferentials[bufferIndexY] = dDdy;

            prd.differential_count = 2u;
        }

        // Each iteration is a segment of the ray path.  The closest hit will
        // return new segments to be traced here.
        for (;;)
        {
            Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
            rtTrace(top_object, ray, prd);

            // Debug quick rendering
            if (limit_to_one_bounce)
            {
                prd.result += prd.attenuation;
                break;
            }

            if (prd.done)
            {
                // We have hit the background or a luminaire
                prd.result += prd.radiance * prd.attenuation;
                break;
            }

            // Russian roulette termination 
            if (prd.depth >= rr_begin_depth)
            {
                float pcont = fmaxf(prd.attenuation);
                if (rnd(prd.seed) >= pcont)
                    break;
                prd.attenuation /= pcont;
            }

            prd.depth++;
            prd.result += prd.radiance * prd.attenuation;

            // Update ray data for the next path segment
            ray_origin = prd.origin;
            ray_direction = prd.direction;
        }

        result += prd.result;
        seed = prd.seed;
    } while (--current_within_pixel_sample);

    //
    // Update the output buffer
    //
    float3 pixel_color = result / (sqrt_num_samples*sqrt_num_samples);

    if (frame_number > 1)
    {
        float a = 1.0f / (float)frame_number;
        float3 old_color = make_float3(output_buffer[launch_index]);
        output_buffer[launch_index] = make_float4(lerp(old_color, pixel_color, a), 1.0f);
    }
    else
    {
        output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
    }
}


//-----------------------------------------------------------------------------
//
//  Emissive surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, emission_color, , );

RT_PROGRAM void diffuseEmitter()
{
    current_prd.radiance = current_prd.countEmitted ? emission_color : make_float3(0.f);
    current_prd.done = true;
}


//-----------------------------------------------------------------------------
//
//  Lambertian surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, diffuse_color, , );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

// Tile determination
rtBuffer<uint, 1>       tileID_to_frameID_buffer;
rtDeclareVariable(uint, frameID, , );
rtDeclareVariable(uint, tileID_path_depth, , );
rtDeclareVariable(uint, virt_tex_size, , );
rtDeclareVariable(uint, tile_size, , );
rtDeclareVariable(uint, tex_mip_id_of_lowest_mip_id_tile, , );
rtDeclareVariable(uint, max_tile_mip_id, , );
rtDeclareVariable(float, distance_to_onscreen_ratio, , );
rtDeclareVariable(float, footprint_scale, , );
rtDeclareVariable(float, footprint_bias, , );

// Path differential variables already defined scene wide (see above).

// From Material
rtTextureSampler<float4, 2> Kd_map;          // classic
rtDeclareVariable(int, texture_atlas, , );   // classic atlased
rtDeclareVariable(int, texture_atlas_0, , ); // classic atlased scaled mipmapped
rtDeclareVariable(int, texture_atlas_1, , ); // classic atlased scaled mipmapped
rtDeclareVariable(int, texture_atlas_2, , ); // classic atlased scaled mipmapped
rtDeclareVariable(int, texture_atlas_3, , ); // classic atlased scaled mipmapped
rtDeclareVariable(int, texture_atlas_4, , ); // classic atlased scaled mipmapped
rtDeclareVariable(int, texture_atlas_5, , ); // classic atlased scaled mipmapped
rtDeclareVariable(int, tile_pool, , );       // virtual atlased
rtDeclareVariable(int, page_table, , );
rtDeclareVariable(float4, texture_atlas_ST, , );
rtDeclareVariable(int, mipID_texture, , );

// Path differentials - Buffer for footprint calculations.
rtBuffer<float2, 3>     footprintDifferentials;

// Path differentials
rtDeclareVariable(float2, T_alpha, attribute T_alpha, ); // TODO(HansCronau): Optimisation - Can be removed by moving calculations to intersection program.
rtDeclareVariable(float2, T_beta, attribute T_beta, );
rtDeclareVariable(float2, T_gamma, attribute T_gamma, );
rtDeclareVariable(float3, non_normalised_normal, attribute non_normalised_normal, ); // TODO(HansCronau): Optimisation - Can be removed by moving calculations to intersection program.
rtDeclareVariable(float3, E_0, attribute E_0, );
rtDeclareVariable(float3, E_2, attribute E_2, );

// Debug - Barycentric coordinates
rtDeclareVariable(float3, P_alpha, attribute P_alpha, );
rtDeclareVariable(float3, P_gamma, attribute P_gamma, );

// Other variables from the intersection program
rtDeclareVariable(float3, texcoord, attribute texcoord, );


static __inline__ __device__ float3 mipToColour(unsigned int mipID)
{
    const int nr_of_colours = 16;
    const float3 colour_ramp[nr_of_colours] = {
        { .99f, 0.0f, 0.0f },
        { .99f, 0.35f, 0.0f },
        { .99f, 0.75f, 0.0f },
        { .99f, .99f, 0.0f },
        { 0.5f, .99f, 0.0f },
        { 0.11f, .99f, 0.0f },
        { 0.0f, .99f, 0.26f },
        { 0.0f, .99f, .99f },
        { 0.0f, 0.61f, .99f },
        { 0.0f, 0.26f, .99f },
        { 0.0f, 0.0f, .99f },
        { 0.4f, 0.0f, .99f },
        { 0.6f, 0.0f, .99f },
        { .99f, 0.0f, .99f },
        { .99f, 0.2f, .99f },
        { .99f, 0.6f, .99f }
    };
    const int ilod = min(mipID, nr_of_colours - 1);
    return colour_ramp[ilod];
}

RT_PROGRAM void diffuse()
{
    float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
    float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);
    float3 hitpoint = ray.origin + t_hit * ray.direction;


    //
    // TRANSFER existing position differentials. (No new differentials are created.)
    // Note: dPdy are included. For both dPdy and dPdx the __dx naming is used.
    //

    const float inv_DN = 1.0f / dot(ray.direction, ffnormal);
    const float3 normal_over_DN = ffnormal * inv_DN; // = ffnormal / dot(ray.direction, ffnormal)
    const float  test_t = dot(hitpoint, ffnormal) * inv_DN; // doesnt work. does this mean that igehy1999 formula 11 does not appy correctly (influencing 12 and 10)?

    for (unsigned int positionDifferential = 0; positionDifferential < current_prd.differential_count; positionDifferential++)
    {
        const uint3 bufferIndex = make_uint3(launch_index, positionDifferential);
        const float3 dPdx = positionDifferentials[bufferIndex];
        const float3 dDdx = directionDifferentials[bufferIndex];
        // Could replace below with function from helpers.h.
        const float dtdx = -dot(dPdx + t_hit * dDdx, normal_over_DN);      // See Igehy1999 formula 12.
        const float3 dPdx2 = (dPdx + t_hit * dDdx) + dtdx * ray.direction; // See Igehy1999 formula 10.
        positionDifferentials[bufferIndex] = dPdx2;
    }


    //
    // COMPUTE FOOTPRINT of path differentials.
    //

    // Calculate barycentric coordinate variables
    // Calculations inspired by Scratchapixel. https://www.scratchapixel.com/lessons/3d-basic-rendering/ray-tracing-rendering-a-triangle/barycentric-coordinates
    // The order of computations was changed for optimisation and easy application to differentials.
    // Correctness L_beta and L_gamma can be checked with barycentric render mode.
    const float  inv_denom = 1.0 / dot(non_normalised_normal, non_normalised_normal);
    const float3 n_inv_denom = non_normalised_normal * inv_denom;
    const float3 L_beta = cross(n_inv_denom, E_2);
    const float3 L_gamma = cross(n_inv_denom, E_0);
    // TODO(HansCronau): Optimisation - Move these calculations to intersection program for optimisation (speed and possibly memory).

    // Calculate footprint differentials (Igehy1999)
    float2 footprint_T = make_float2(0.f);
    for (unsigned int positionDifferential = 0; positionDifferential < current_prd.differential_count; positionDifferential++)
    {
        const uint3 bufferIndex = make_uint3(launch_index, positionDifferential);
        const float3 dPdx_world = positionDifferentials[bufferIndex];

        const float3 dPdx_object = rtTransformVector(RT_WORLD_TO_OBJECT, dPdx_world);

        // Calculate barycentric coordinates.
        const float beta = dot(L_beta, dPdx_object);
        const float gamma = dot(L_gamma, dPdx_object);

        // Calculate differential as one would calculate ordinary UV coordinates,
        // but make values relative by subtracting a corner vector's position.
        const float2 dTdx = beta*(T_beta - T_alpha) + gamma*(T_gamma - T_alpha); // + (1.0f - beta - gamma)*(T_alpha - T_alpha) is left out because it equals 0.
        // Vector dTdx represents a difference, but can be both positive or negative.
        // Make sure all dTdx are expressed positively. (All vectors should point up in Fig. 3(b), Suykens2001.)
        const float2 dTdx_pos = dTdx.y > 0.0f ? dTdx : -dTdx;

        // Store footprint differential for processing (see next step).
        footprintDifferentials[bufferIndex] = dTdx_pos;
        // Add footprint differential to sum of all footprints.
        footprint_T += dTdx_pos;
    }

    // Process footprint differentials
    const float2 footprint_PT = make_float2(footprint_T.y, -footprint_T.x); // PT is any vector perpendicular to T.
    float2 footprint_DA = make_float2(0);
    for (unsigned int positionDifferential = 0; positionDifferential < current_prd.differential_count; positionDifferential++)
    {
        const uint3 bufferIndex = make_uint3(launch_index, positionDifferential);
        const float2 dTdx = footprintDifferentials[bufferIndex];
        if (dot(footprint_PT, dTdx) > 0)
        {
            footprint_DA += dTdx;
        }
    }
    float2 footprint_DB = footprint_T - footprint_DA;


    //
    // CHOOSE DELTA
    //

    const unsigned int num_samples = sqrt_num_samples*sqrt_num_samples;                          // = N (Suykens2001)
    const float suykens_delta = 1.0f / powf(num_samples, 1.0f / current_prd.differential_count); // Is $1/\sqrt[M]{N}$ (Suykens2001)
    const float delta = 2.0f * suykens_delta;                                                    // Because U and V both span only half the respective width and height of the screen. (See inv_screen_times_two)

    // Scale footprint from UV space to texture's texel space (correct for atlas scale) and apply delta.
    const uint virt_tex_size_in_tiles = virt_tex_size / tile_size;
    const float2 texture_atlas_S = make_float2(texture_atlas_ST.x, texture_atlas_ST.y);

    // Combine scaling effects: delta (Suykens2001) * subtexture to atlas (scales down differentials) * custom scaling (from host device)
    const float2 combined_footprint_scale = delta * texture_atlas_S * footprint_scale;
    // NOTE(HansCronau): Optimisation possible if footprints are only used for virtual textures: move multiplication by tile_size to line above.

    // Calculate texture differentials from footprint vectors.
    const float2 dTdx_A = combined_footprint_scale * footprint_DA + footprint_bias * normalize(footprint_DA);
    const float2 dTdx_B = combined_footprint_scale * footprint_DB + footprint_bias * normalize(footprint_DB);
    // NOTE(HansCronau): Optimisation possible if bias is 0: remove bias calculations.

    // Upscale differentials when sampling from textures representing tile data per texel (i.e. mipID texture and page table texture).
    const float2 dTiledx_A = tile_size * dTdx_A; // = texel mipID to tile mipID * differential
    const float2 dTiledx_B = tile_size * dTdx_B;

    //
    // WRITE TO TILEID BUFFER (VIRTUAL TEXTURE TILE DETERMINATION)
    //

    // Scale and translate UV coordinates from subtexture space to atlas texture space. Modulo is to support tiling textures.
    float atlased_u = positive_modulo(texcoord.x, 1.0f) * texture_atlas_ST.x + texture_atlas_ST.z;
    float atlased_v = positive_modulo(texcoord.y, 1.0f) * texture_atlas_ST.y + texture_atlas_ST.w;

    // If current path hasn't had too many bounces, we have room in our buffer to record hit tileIDs.
    uint tile_mipID_calc, tile_mipID_tex, tile_mipID, tile_x, tile_y;
    if (current_prd.depth < tileID_path_depth) {
        // Determine index within tileID buffer to write to.
        uint3 bufferIdx = make_uint3(launch_index, current_prd.within_fragment_path_id * tileID_path_depth + current_prd.depth); // nog checken voor randgevallen

        // Determine mipID. (MipID is an inverse mipmap level so that 2^mipID yields mip level's dimensions.)

        // Option A: Calculate mipID from differentials. (Chajdas2010, OpenGL 4.6 8.14.1)
        const float mipCalcHack = 5.5f; // TODO(HansCronau): Remove hack. Don't know why it is required. Expected same results as when sampling from mipID texture.
        const int tile_mip_level = mipCalcHack + log2(max(length(dTiledx_A), length(dTiledx_B))); // Note: rounding down mip level by cast to int increases mip resolution.
        tile_mipID_calc = max_tile_mip_id - clamp(tile_mip_level, 0, static_cast<int>(max_tile_mip_id));

        // Option B: Sample mipID from mipID texture using differentials.
        // This implicitly caps the mipID to the max mip ID.
        tile_mipID_tex = rtTex2DGrad<uint>(mipID_texture, atlased_u, atlased_v, dTiledx_A, dTiledx_B);

        // Option C: Calculate mipID based on path distance.
        // Keep track of the total distance traveled by the path.
        //current_prd.total_path_distance += t_hit;
        //uint tex_mipID = ceil(log2f(virt_tex_size / (current_prd.total_path_distance * distance_to_onscreen_ratio)));
        //tile_mipID = clamp(tex_mipID - tex_mip_id_of_lowest_mip_id_tile), 0, max_tile_mip_id);

        tile_mipID = tile_mipID_calc;

        // Calculate remaining TileID x and y.
        int mipWidth = dimensionsForMipID(tile_mipID);
        tile_x = atlased_u * mipWidth;
        tile_y = atlased_v * mipWidth;

        // Find the corresponding tile index in the tileID_to_frameID_buffer.
        uint tileID_index = offsetForMipID(tile_mipID) + mipWidth * tile_y + tile_x;

        // Store current frameID at that index.
        tileID_to_frameID_buffer[tileID_index] = frameID;
    }
    else
    {
        // Warning: values below are invalid when 1 or higher. (Your graphics driver may crash.)
        tile_x = 0;
        tile_y = 0;
        tile_mipID = 0;
    }


    //
    // SHADE
    // Note: f/pdf = 1 since we are perfectly importance sampling lambertian with cosine density.
    //

    // NOTE(HansCronau): For debugging purposes here comes a big nasty switch statement.

    // Variable names used in multiple switch statement cases must be defined outside of it:
    float3 scaleAndBias;
    float3 colour;

    switch (debug_render_mode)
    {
    case 0:
        // Render diffuse single colour
        colour = make_float3(t_hit / 10000.0f);
        current_prd.attenuation = current_prd.attenuation * colour;
        break;
    case 1:
        // Render barycentric coordinates
        const float3 P_intersection = rtTransformVector(RT_WORLD_TO_OBJECT, hitpoint);
        const float beta = dot(L_beta, P_intersection - P_gamma);
        const float gamma = dot(L_gamma, P_intersection - P_alpha);
        colour = make_float3(1 - beta - gamma, beta, gamma);
        current_prd.attenuation = current_prd.attenuation * colour;
        break;
    case 2:
        // Render interpolated UV
        colour = fminf(texcoord, make_float3(.99, .99, .99));
        current_prd.attenuation = current_prd.attenuation * colour;
        break;
    case 3:
        // Render diffuse with classic texture
        colour = make_float3(tex2D(Kd_map, texcoord.x, texcoord.y)); // Standard texture sampling.
        current_prd.attenuation = current_prd.attenuation * colour;
        break;
    case 4:
        // Render atlased interpolated UV
        colour = make_float3(atlased_u, atlased_v, 0);
        current_prd.attenuation = current_prd.attenuation * colour;
        break;
    case 5:
        // Render diffuse with classic texture atlas
        colour = make_float3(rtTex2D<float4>(texture_atlas, atlased_u, atlased_v));
        current_prd.attenuation = current_prd.attenuation * colour;
        break;
    case 6:
        // Render diffuse with classic, scaled texture atlas
        colour = make_float3(rtTex2D<float4>(texture_atlas_5, atlased_u, atlased_v));
        current_prd.attenuation = current_prd.attenuation * colour;
        break;
    case 7:
        // Render diffuse with classic, scaled, and mipmapped texture atlas
        if (tile_mipID == 0)
        {
            colour = make_float3(rtTex2D<float4>(texture_atlas_0, atlased_u, atlased_v));
        }
        else if (tile_mipID == 1)
        {
            colour = make_float3(rtTex2D<float4>(texture_atlas_1, atlased_u, atlased_v));
        }
        else if (tile_mipID == 2)
        {
            colour = make_float3(rtTex2D<float4>(texture_atlas_2, atlased_u, atlased_v));
        }
        else if (tile_mipID == 3)
        {
            colour = make_float3(rtTex2D<float4>(texture_atlas_3, atlased_u, atlased_v));
        }
        else if (tile_mipID == 4)
        {
            colour = make_float3(rtTex2D<float4>(texture_atlas_4, atlased_u, atlased_v));
        }
        else
        {
            colour = make_float3(rtTex2D<float4>(texture_atlas_5, atlased_u, atlased_v));
        }
        current_prd.attenuation = current_prd.attenuation * colour;
        break;
    case 8:
        // Render differential footprint
        colour = make_float3(footprint_T.x, -footprint_T.x, footprint_T.y);
        colour = clamp(colour, 0, .99);
        current_prd.attenuation = current_prd.attenuation * colour;
        break;
    case 9:
        // Render atlased differential footprint
        colour = make_float3(texture_atlas_ST.x * footprint_T.x, -texture_atlas_ST.x * footprint_T.x, texture_atlas_ST.y * footprint_T.y);
        colour = clamp(colour, 0, .99);
        current_prd.attenuation = current_prd.attenuation * colour;
        break;
    case 10:
        // Render atlased differential footprint A component
        colour = make_float3(texture_atlas_ST.x * dTdx_A.x, texture_atlas_ST.x * -dTdx_A.x, texture_atlas_ST.x * dTdx_A.y);
        colour = clamp(colour, 0, .99);
        current_prd.attenuation = current_prd.attenuation * colour;
        break;
    case 11:
        // Render atlased differential footprint B component
        colour = make_float3(texture_atlas_ST.y * dTdx_B.x, texture_atlas_ST.y * -dTdx_B.x, texture_atlas_ST.y * dTdx_B.y);
        colour = clamp(colour, 0, .99);
        current_prd.attenuation = current_prd.attenuation * colour;
        break;
    case 12:
        // Render mipID colours sampling mipID from texture
        colour = mipToColour(tile_mipID_tex);
        current_prd.attenuation = current_prd.attenuation * colour;
        break;
    case 13:
        // Render mipID colours calculating mipID from differentials
        colour = mipToColour(tile_mipID_calc);
        current_prd.attenuation = current_prd.attenuation * colour;
        break;
    case 14:
        // Render TileID
        if (tile_mipID == 0)
        {
            // MipID 0 corresponds to the colour black (0,0,0), which does not render well. Use grey instead.
            colour = make_float3(.01, .01, .01);
        }
        else
        {
            colour = make_float3(
                (float)tile_mipID / mipIDForDimensions(virt_tex_size_in_tiles),
                (float)tile_x / virt_tex_size_in_tiles,
                (float)tile_y / virt_tex_size_in_tiles
                );
        }
        current_prd.attenuation = current_prd.attenuation * colour;
        break;
    case 15:
        // Render scaleAndBias.
        //scaleAndBias = make_float3(rtTex2DLod<float4>(page_table, texcoord.x, texcoord.y, 0.0)); // Sample from page table texture.
        scaleAndBias = make_float3(rtTex2DGrad<float4>(page_table, atlased_u, atlased_v, dTiledx_A, dTiledx_B)); // Sample from page table texture.
        colour.x = scaleAndBias.x / (virt_tex_size / tile_size);
        colour.y = scaleAndBias.y;
        colour.z = scaleAndBias.z;
        current_prd.attenuation = current_prd.attenuation * colour;
        break;
    case 16:
        // Render tile pool directly.
        colour = make_float3(rtTex2D<float4>(tile_pool, atlased_u, atlased_v)); // Direct tile pool sampling.
        current_prd.attenuation = current_prd.attenuation * colour;
        break;
    case 17:
    default:
        // Render diffuse with virtual texture.
        scaleAndBias = make_float3(rtTex2DGrad<float4>(page_table, atlased_u, atlased_v, dTiledx_A, dTiledx_B)); // Sample from page table texture.
        float2 physicalAddress = devirtualiseAddress(scaleAndBias, make_float2(atlased_u, atlased_v)); // Translate virtual to physical address.
        colour = make_float3(rtTex2D<float4>(tile_pool, physicalAddress.x, physicalAddress.y)); // Virtual texture sampling.
        current_prd.attenuation = current_prd.attenuation * colour;
        break;
    }

    current_prd.countEmitted = false;


    //
    // SCATTER
    // Generate a reflection ray.  This will be traced back in ray-gen.
    //

    current_prd.origin = hitpoint;

    // Create some random variables between 0 and 1.
    float z1 = rnd(current_prd.seed);
    float z2 = rnd(current_prd.seed);

    // Create an orthonormal basis (ONB).
    optix::Onb onb(ffnormal);

    // If using differentials and room left in differential buffer:
    if (current_prd.differential_count < differentialDepth * 2)
    {
        // Sample a point and differentials on a cosine weighted hemisphere.
        float3 p, dpdz1, dpdz2;
        cosine_sample_hemisphere_incl_differentials(z1, z2, p, dpdz1, dpdz2);

        // Project the point on the hemisphere and the differentials onto the orthonormal basis.
        current_prd.direction = onb.m_tangent * p.x + onb.m_binormal * p.y + ffnormal * p.z;
        const float3 dDdx = onb.m_tangent * dpdz1.x + onb.m_binormal * dpdz1.y + ffnormal * dpdz1.z;
        const float3 dDdy = onb.m_tangent * dpdz2.x + onb.m_binormal * dpdz2.y + ffnormal * dpdz2.z;

        // Create new position differentials.
        const float3 dPdx = make_float3(0);
        const float3 dPdy = make_float3(0);

        // Determine index within differential buffers.
        const uint3 differentialBufferIndexX = make_uint3(launch_index, current_prd.differential_count + 0);
        const uint3 differentialBufferIndexY = make_uint3(launch_index, current_prd.differential_count + 1);

        // Add new differentials to differential buffers.
        positionDifferentials[differentialBufferIndexX] = dPdx;
        positionDifferentials[differentialBufferIndexY] = dPdy;
        directionDifferentials[differentialBufferIndexX] = dDdx;
        directionDifferentials[differentialBufferIndexY] = dDdy;
        current_prd.differential_count += 2u;
    }
    else // Classic diffuse scatter without calculating differentials.
    {
        // Sample a point and differentials on a cosine weighted hemisphere.
        float3 p;
        cosine_sample_hemisphere(z1, z2, p);

        // Project the point on the hemisphere onto the orthonormal basis.
        // Equivalent to current_prd.direction = tangent * p.x + binormal * p.y + ffnormal * p.z;
        // Binormal is graphics lingo for bitangent.
        onb.inverse_transform(p);
        current_prd.direction = p;
    }


    //
    // NEXT EVENT ESTIMATION
    // Compute direct lighting.
    //

    unsigned int num_lights = lights.size();
    float3 result = make_float3(0.0f);

    for (int i = 0; i < num_lights; ++i)
    {
        // Choose random point on light
        ParallelogramLight light = lights[i];
        const float z1 = rnd(current_prd.seed);
        const float z2 = rnd(current_prd.seed);
        const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

        // Calculate properties of light sample (for area based pdf)
        const float  Ldist = length(light_pos - hitpoint);
        const float3 L = normalize(light_pos - hitpoint);
        const float  nDl = dot(ffnormal, L);
        const float  LnDl = dot(light.normal, L);

        // cast shadow ray
        if (nDl > 0.0f && LnDl > 0.0f)
        {
            PerRayData_pathtrace_shadow shadow_prd;
            shadow_prd.inShadow = false;
            // Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
            Ray shadow_ray = make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
            rtTrace(top_object, shadow_ray, shadow_prd);

            if (!shadow_prd.inShadow)
            {
                const float A = length(cross(light.v1, light.v2));
                // convert area based pdf to solid angle
                const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
                result += light.emission * weight;
            }
        }
    }

    current_prd.radiance = result;
}


// This closest hit program is an adapted copy-paste of the above diffuse() program.
// Code was removed so that it runs only the (non-virtual) classic, scaled texture atlas render mode.
RT_PROGRAM void atlased_diffuse()
{
    float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
    float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);
    float3 hitpoint = ray.origin + t_hit * ray.direction;

    //
    // SHADE
    // Note: f/pdf = 1 since we are perfectly importance sampling lambertian with cosine density.
    //

    // Scale and translate UV coordinates from subtexture space to atlas texture space. Modulo is to support tiling textures.
    float atlased_u = positive_modulo(texcoord.x, 1.0f) * texture_atlas_ST.x + texture_atlas_ST.z;
    float atlased_v = positive_modulo(texcoord.y, 1.0f) * texture_atlas_ST.y + texture_atlas_ST.w;

    // Render diffuse with classic texture atlas
    float3 colour = make_float3(rtTex2D<float4>(texture_atlas, atlased_u, atlased_v));
    current_prd.attenuation = current_prd.attenuation * colour;
    current_prd.countEmitted = false;


    //
    // SCATTER
    // Generate a reflection ray.  This will be traced back in ray-gen.
    //

    current_prd.origin = hitpoint;

    // Create some random variables between 0 and 1.
    float z1 = rnd(current_prd.seed);
    float z2 = rnd(current_prd.seed);

    // Create an orthonormal basis (ONB).
    optix::Onb onb(ffnormal);

    // Classic diffuse scatter without calculating differentials.
    // Sample a point and differentials on a cosine weighted hemisphere.
    float3 p;
    cosine_sample_hemisphere(z1, z2, p);

    // Project the point on the hemisphere onto the orthonormal basis.
    // Equivalent to current_prd.direction = tangent * p.x + binormal * p.y + ffnormal * p.z;
    // Binormal is graphics lingo for bitangent.
    onb.inverse_transform(p);
    current_prd.direction = p;


    //
    // NEXT EVENT ESTIMATION
    // Compute direct lighting.
    //

    unsigned int num_lights = lights.size();
    float3 result = make_float3(0.0f);

    for (int i = 0; i < num_lights; ++i)
    {
        // Choose random point on light
        ParallelogramLight light = lights[i];
        const float z1 = rnd(current_prd.seed);
        const float z2 = rnd(current_prd.seed);
        const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

        // Calculate properties of light sample (for area based pdf)
        const float  Ldist = length(light_pos - hitpoint);
        const float3 L = normalize(light_pos - hitpoint);
        const float  nDl = dot(ffnormal, L);
        const float  LnDl = dot(light.normal, L);

        // cast shadow ray
        if (nDl > 0.0f && LnDl > 0.0f)
        {
            PerRayData_pathtrace_shadow shadow_prd;
            shadow_prd.inShadow = false;
            // Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
            Ray shadow_ray = make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
            rtTrace(top_object, shadow_ray, shadow_prd);

            if (!shadow_prd.inShadow)
            {
                const float A = length(cross(light.v1, light.v2));
                // convert area based pdf to solid angle
                const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
                result += light.emission * weight;
            }
        }
    }

    current_prd.radiance = result;
}


// This closest hit program is an adapted copy-paste of the above diffuse() program.
// Code was removed so that it runs only the virtual texture render mode.
RT_PROGRAM void virtual_diffuse()
{
    float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
    float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);
    float3 hitpoint = ray.origin + t_hit * ray.direction;


    //
    // TRANSFER existing position differentials. (No new differentials are created.)
    // Note: dPdy are included. For both dPdy and dPdx the __dx naming is used.
    //

    const float inv_DN = 1.0f / dot(ray.direction, ffnormal);
    const float3 normal_over_DN = ffnormal * inv_DN; // = ffnormal / dot(ray.direction, ffnormal)
    const float  test_t = dot(hitpoint, ffnormal) * inv_DN; // doesnt work. does this mean that igehy1999 formula 11 does not appy correctly (influencing 12 and 10)?

    for (unsigned int positionDifferential = 0; positionDifferential < current_prd.differential_count; positionDifferential++)
    {
        const uint3 bufferIndex = make_uint3(launch_index, positionDifferential);
        const float3 dPdx = positionDifferentials[bufferIndex];
        const float3 dDdx = directionDifferentials[bufferIndex];
        // Could replace below with function from helpers.h.
        const float dtdx = -dot(dPdx + t_hit * dDdx, normal_over_DN);      // See Igehy1999 formula 12.
        const float3 dPdx2 = (dPdx + t_hit * dDdx) + dtdx * ray.direction; // See Igehy1999 formula 10.
        positionDifferentials[bufferIndex] = dPdx2;
    }


    //
    // COMPUTE FOOTPRINT of path differentials.
    //

    // Calculate barycentric coordinate variables
    // Calculations inspired by Scratchapixel. https://www.scratchapixel.com/lessons/3d-basic-rendering/ray-tracing-rendering-a-triangle/barycentric-coordinates
    // The order of computations was changed for optimisation and easy application to differentials.
    // Correctness L_beta and L_gamma can be checked with barycentric render mode.
    const float  inv_denom = 1.0 / dot(non_normalised_normal, non_normalised_normal);
    const float3 n_inv_denom = non_normalised_normal * inv_denom;
    const float3 L_beta = cross(n_inv_denom, E_2);
    const float3 L_gamma = cross(n_inv_denom, E_0);
    // TODO(HansCronau): Optimisation - Move these calculations to intersection program for optimisation (speed and possibly memory).

    // Calculate footprint differentials (Igehy1999)
    float2 footprint_T = make_float2(0.f);
    for (unsigned int positionDifferential = 0; positionDifferential < current_prd.differential_count; positionDifferential++)
    {
        const uint3 bufferIndex = make_uint3(launch_index, positionDifferential);
        const float3 dPdx_world = positionDifferentials[bufferIndex];

        const float3 dPdx_object = rtTransformVector(RT_WORLD_TO_OBJECT, dPdx_world);

        // Calculate barycentric coordinates.
        const float beta = dot(L_beta, dPdx_object);
        const float gamma = dot(L_gamma, dPdx_object);

        // Calculate differential as one would calculate ordinary UV coordinates,
        // but make values relative by subtracting a corner vector's position.
        const float2 dTdx = beta*(T_beta - T_alpha) + gamma*(T_gamma - T_alpha); // + (1.0f - beta - gamma)*(T_alpha - T_alpha) is left out because it equals 0.
        // Vector dTdx represents a difference, but can be both positive or negative.
        // Make sure all dTdx are expressed positively. (All vectors should point up in Fig. 3(b), Suykens2001.)
        const float2 dTdx_pos = dTdx.y > 0.0f ? dTdx : -dTdx;

        // Store footprint differential for processing (see next step).
        footprintDifferentials[bufferIndex] = dTdx_pos;
        // Add footprint differential to sum of all footprints.
        footprint_T += dTdx_pos;
    }

    // Process footprint differentials
    const float2 footprint_PT = make_float2(footprint_T.y, -footprint_T.x); // PT is any vector perpendicular to T.
    float2 footprint_DA = make_float2(0);
    for (unsigned int positionDifferential = 0; positionDifferential < current_prd.differential_count; positionDifferential++)
    {
        const uint3 bufferIndex = make_uint3(launch_index, positionDifferential);
        const float2 dTdx = footprintDifferentials[bufferIndex];
        if (dot(footprint_PT, dTdx) > 0)
        {
            footprint_DA += dTdx;
        }
    }
    float2 footprint_DB = footprint_T - footprint_DA;


    //
    // CHOOSE DELTA
    //

    const unsigned int num_samples = sqrt_num_samples*sqrt_num_samples;                          // = N (Suykens2001)
    const float suykens_delta = 1.0f / powf(num_samples, 1.0f / current_prd.differential_count); // Is $1/\sqrt[M]{N}$ (Suykens2001)
    const float delta = 2.0f * suykens_delta;                                                    // Because U and V both span only half the respective width and height of the screen. (See inv_screen_times_two)

    // Scale footprint from UV space to texture's texel space (correct for atlas scale) and apply delta.
    const uint virt_tex_size_in_tiles = virt_tex_size / tile_size;
    const float2 texture_atlas_S = make_float2(texture_atlas_ST.x, texture_atlas_ST.y);

    // Combine scaling effects: delta (Suykens2001) * subtexture to atlas (scales down differentials) * custom scaling (from host device)
    const float2 combined_footprint_scale = delta * texture_atlas_S * footprint_scale;
    // NOTE(HansCronau): Optimisation possible if footprints are only used for virtual textures: move multiplication by tile_size to line above.

    // Calculate texture differentials from footprint vectors.
    const float2 dTdx_A = combined_footprint_scale * footprint_DA + footprint_bias * normalize(footprint_DA);
    const float2 dTdx_B = combined_footprint_scale * footprint_DB + footprint_bias * normalize(footprint_DB);
    // NOTE(HansCronau): Optimisation possible if bias is 0: remove bias calculations.

    // Upscale differentials when sampling from textures representing tile data per texel (i.e. mipID texture and page table texture).
    const float2 dTiledx_A = tile_size * dTdx_A; // = texel mipID to tile mipID * differential
    const float2 dTiledx_B = tile_size * dTdx_B;

    //
    // WRITE TO TILEID BUFFER (VIRTUAL TEXTURE TILE DETERMINATION)
    //

    // Scale and translate UV coordinates from subtexture space to atlas texture space. Modulo is to support tiling textures.
    float atlased_u = positive_modulo(texcoord.x, 1.0f) * texture_atlas_ST.x + texture_atlas_ST.z;
    float atlased_v = positive_modulo(texcoord.y, 1.0f) * texture_atlas_ST.y + texture_atlas_ST.w;

    // If current path hasn't had too many bounces, we have room in our buffer to record hit tileIDs.
    uint tile_mipID_calc, tile_mipID_tex, tile_mipID, tile_x, tile_y;
    if (current_prd.depth < tileID_path_depth) {
        // Determine index within tileID buffer to write to.
        uint3 bufferIdx = make_uint3(launch_index, current_prd.within_fragment_path_id * tileID_path_depth + current_prd.depth); // nog checken voor randgevallen

        // Determine mipID. (MipID is an inverse mipmap level so that 2^mipID yields mip level's dimensions.)

        // Option A: Calculate mipID from differentials. (Chajdas2010, OpenGL 4.6 8.14.1)
        const float mipCalcHack = 5.5f; // TODO(HansCronau): Remove hack. Don't know why it is required. Expected same results as when sampling from mipID texture.
        const int tile_mip_level = mipCalcHack + log2(max(length(dTiledx_A), length(dTiledx_B))); // Note: rounding down mip level by cast to int increases mip resolution.
        tile_mipID_calc = max_tile_mip_id - clamp(tile_mip_level, 0, static_cast<int>(max_tile_mip_id));

        // Option B: Sample mipID from mipID texture using differentials.
        // This implicitly caps the mipID to the max mip ID.
        //tile_mipID_tex = rtTex2DGrad<uint>(mipID_texture, atlased_u, atlased_v, dTiledx_A, dTiledx_B);

        // Option C: Calculate mipID based on path distance.
        // Keep track of the total distance traveled by the path.
        //current_prd.total_path_distance += t_hit;
        //uint tex_mipID = ceil(log2f(virt_tex_size / (current_prd.total_path_distance * distance_to_onscreen_ratio)));
        //tile_mipID = clamp(tex_mipID - tex_mip_id_of_lowest_mip_id_tile), 0, max_tile_mip_id);

        tile_mipID = tile_mipID_calc;

        // Calculate remaining TileID x and y.
        int mipWidth = dimensionsForMipID(tile_mipID);
        tile_x = atlased_u * mipWidth;
        tile_y = atlased_v * mipWidth;

        // Find the corresponding tile index in the tileID_to_frameID_buffer.
        uint tileID_index = offsetForMipID(tile_mipID) + mipWidth * tile_y + tile_x;

        // Store current frameID at that index.
        tileID_to_frameID_buffer[tileID_index] = frameID;
    }
    else
    {
        // Warning: values below are invalid when 1 or higher. (Your graphics driver may crash.)
        tile_x = 0;
        tile_y = 0;
        tile_mipID = 0;
    }


    //
    // SHADE
    // Note: f/pdf = 1 since we are perfectly importance sampling lambertian with cosine density.
    //

    // Render diffuse with virtual texture.
    float3 scaleAndBias = make_float3(rtTex2DGrad<float4>(page_table, atlased_u, atlased_v, dTiledx_A, dTiledx_B)); // Sample from page table texture.
    float2 physicalAddress = devirtualiseAddress(scaleAndBias, make_float2(atlased_u, atlased_v)); // Translate virtual to physical address.
    float3 colour = make_float3(rtTex2D<float4>(tile_pool, physicalAddress.x, physicalAddress.y)); // Virtual texture sampling.
    current_prd.attenuation = current_prd.attenuation * colour;
    current_prd.countEmitted = false;


    //
    // SCATTER
    // Generate a reflection ray.  This will be traced back in ray-gen.
    //

    current_prd.origin = hitpoint;

    // Create some random variables between 0 and 1.
    float z1 = rnd(current_prd.seed);
    float z2 = rnd(current_prd.seed);

    // Create an orthonormal basis (ONB).
    optix::Onb onb(ffnormal);

    // If using differentials and room left in differential buffer:
    if (current_prd.differential_count < differentialDepth * 2)
    {
        // Sample a point and differentials on a cosine weighted hemisphere.
        float3 p, dpdz1, dpdz2;
        cosine_sample_hemisphere_incl_differentials(z1, z2, p, dpdz1, dpdz2);

        // Project the point on the hemisphere and the differentials onto the orthonormal basis.
        current_prd.direction = onb.m_tangent * p.x + onb.m_binormal * p.y + ffnormal * p.z;
        const float3 dDdx = onb.m_tangent * dpdz1.x + onb.m_binormal * dpdz1.y + ffnormal * dpdz1.z;
        const float3 dDdy = onb.m_tangent * dpdz2.x + onb.m_binormal * dpdz2.y + ffnormal * dpdz2.z;

        // Create new position differentials.
        const float3 dPdx = make_float3(0);
        const float3 dPdy = make_float3(0);

        // Determine index within differential buffers.
        const uint3 differentialBufferIndexX = make_uint3(launch_index, current_prd.differential_count + 0);
        const uint3 differentialBufferIndexY = make_uint3(launch_index, current_prd.differential_count + 1);

        // Add new differentials to differential buffers.
        positionDifferentials[differentialBufferIndexX] = dPdx;
        positionDifferentials[differentialBufferIndexY] = dPdy;
        directionDifferentials[differentialBufferIndexX] = dDdx;
        directionDifferentials[differentialBufferIndexY] = dDdy;
        current_prd.differential_count += 2u;
    }
    else // Classic diffuse scatter without calculating differentials.
    {
        // Sample a point and differentials on a cosine weighted hemisphere.
        float3 p;
        cosine_sample_hemisphere(z1, z2, p);

        // Project the point on the hemisphere onto the orthonormal basis.
        // Equivalent to current_prd.direction = tangent * p.x + binormal * p.y + ffnormal * p.z;
        // Binormal is graphics lingo for bitangent.
        onb.inverse_transform(p);
        current_prd.direction = p;
    }


    //
    // NEXT EVENT ESTIMATION
    // Compute direct lighting.
    //

    unsigned int num_lights = lights.size();
    float3 result = make_float3(0.0f);

    for (int i = 0; i < num_lights; ++i)
    {
        // Choose random point on light
        ParallelogramLight light = lights[i];
        const float z1 = rnd(current_prd.seed);
        const float z2 = rnd(current_prd.seed);
        const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

        // Calculate properties of light sample (for area based pdf)
        const float  Ldist = length(light_pos - hitpoint);
        const float3 L = normalize(light_pos - hitpoint);
        const float  nDl = dot(ffnormal, L);
        const float  LnDl = dot(light.normal, L);

        // cast shadow ray
        if (nDl > 0.0f && LnDl > 0.0f)
        {
            PerRayData_pathtrace_shadow shadow_prd;
            shadow_prd.inShadow = false;
            // Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
            Ray shadow_ray = make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
            rtTrace(top_object, shadow_ray, shadow_prd);

            if (!shadow_prd.inShadow)
            {
                const float A = length(cross(light.v1, light.v2));
                // convert area based pdf to solid angle
                const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
                result += light.emission * weight;
            }
        }
    }

    current_prd.radiance = result;
}


//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void shadow()
{
    current_prd_shadow.inShadow = true;
    rtTerminateRay();
}


//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
#ifdef VT_DEBUG_OPTIX_PRINT_ENABLED
    rtPrintf("Exception!\n");
#endif
    output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, bg_color, , );

RT_PROGRAM void miss()
{
#ifdef VT_DEBUG_OPTIX_PRINT_ENABLED
    rtPrintf("Miss.\n");
#endif
    current_prd.radiance = bg_color;
    current_prd.done = true;
}
