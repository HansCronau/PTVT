#include "hip/hip_runtime.h"
/*
 * Path Traced Virtual Textures (PTVT)
 * Copyright 2018 Hans Cronau
 *
 * File based on the Optix SDK optixPathTracer sample,
 * Copyright 2016 Nvidia Corporation
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

// This is to be plugged into an RTgeometry object to represent
// a triangle mesh with a vertex buffer of triangle soup (triangle list)
// with an interleaved position, normal, texturecoordinate layout.

rtBuffer<float3> vertex_buffer;     
rtBuffer<float3> normal_buffer;
rtBuffer<float2> texcoord_buffer;
rtBuffer<int3>   index_buffer;
rtBuffer<int>    material_buffer;

rtDeclareVariable(float3, texcoord,         attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 

rtDeclareVariable(float3, back_hit_point,   attribute back_hit_point, ); 
rtDeclareVariable(float3, front_hit_point,  attribute front_hit_point, ); 

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

// Path differentials
rtDeclareVariable(float2, T_alpha, attribute T_alpha, );
rtDeclareVariable(float2, T_beta, attribute T_beta, );
rtDeclareVariable(float2, T_gamma, attribute T_gamma, );
rtDeclareVariable(float3, non_normalised_normal, attribute non_normalised_normal, );
rtDeclareVariable(float3, E_0, attribute E_0, );
rtDeclareVariable(float3, E_2, attribute E_2, );

// Debug - Barycentric coordinates
rtDeclareVariable(float3, P_alpha, attribute P_alpha, );
rtDeclareVariable(float3, P_gamma, attribute P_gamma, );

// Legenda variables:
// corners triangle:                                   p0,    p1,   p2
// opposing edges:                                            e1,   e0
// corresponding barycentric coordinates intersection: alpha, beta, gamma
// cartesian coordinates intersection (object space):          i
static __device__ bool intersect_triangle_branchless(const Ray&    ray,
                                                     const float3& p0,
                                                     const float3& p1,
													 const float3& p2,
													 const float3& e0,
													 const float3& e1,
                                                           float3& n,
                                                           float&  t,
                                                           float&  beta,
                                                           float&  gamma)
{
  n  = cross( e1, e0 );

  const float3 m = ( 1.0f / dot( n, ray.direction ) ) * ( p0 - ray.origin );
  const float3 i = cross(ray.direction, m);

  beta  = dot( i, e1 );
  gamma = dot( i, e0 );
  t     = dot( n, m );

  return ( (t<ray.tmax) & (t>ray.tmin) & (beta>=0.0f) & (gamma>=0.0f) & (beta+gamma<=1) );
}

RT_PROGRAM void mesh_intersect(int primIdx)
{
	const int3 v_idx = index_buffer[primIdx];

	const float3 p0 = vertex_buffer[ v_idx.x ];
	const float3 p1 = vertex_buffer[ v_idx.y ];
	const float3 p2 = vertex_buffer[ v_idx.z ];

	// Will be used for triangle intersection and differential calculation.
	const float3 e0 = p1 - p0;
    const float3 e1 = p0 - p2;
    // Note: e0 and e1 are named by OptiX convention. They correspond to E_0 and E_2 respectively.
	
	// Intersect ray with triangle.
	float3 n;
	float  t, beta, gamma;
	if (intersect_triangle_branchless(ray, p0, p1, p2, e0, e1, n, t, beta, gamma)) {

		if(  rtPotentialIntersection( t ) ) {

			// Normals
			geometric_normal = normalize( n );
			if( normal_buffer.size() == 0 ) {
			shading_normal = geometric_normal; 
			} else {
			float3 n0 = normal_buffer[ v_idx.x ];
			float3 n1 = normal_buffer[ v_idx.y ];
			float3 n2 = normal_buffer[ v_idx.z ];
			shading_normal = normalize( n1*beta + n2*gamma + n0*(1.0f-beta-gamma) );
			}

			// Differentials - Texture coordinates
			if (texcoord_buffer.size() == 0) {
				T_alpha  = make_float2( 0.0f, 0.0f );
				T_beta   = make_float2( 0.0f, 0.0f );
				T_gamma  = make_float2( 0.0f, 0.0f );
				texcoord = make_float3( 0.0f, 0.0f, 0.0f ); // OptiX code has UV-coords in R3.
			}
			else {
				T_alpha  = texcoord_buffer[ v_idx.x ];
				T_beta   = texcoord_buffer[ v_idx.y ];
				T_gamma  = texcoord_buffer[ v_idx.z ];
				texcoord = make_float3(T_beta*beta + T_gamma*gamma + T_alpha*(1.0f - beta - gamma));
			}

			// Differentials - Edges
            non_normalised_normal = n;
			E_0 = e0; // = p1 - p0
			E_2 = e1; // = p0 - p2

            // Debug - Barycentric coordinates
            P_alpha = p0;
            P_gamma = p2;

			rtReportIntersection(material_buffer[primIdx]);
		}
	}
}

RT_PROGRAM void mesh_bounds (int primIdx, float result[6])
{
  const int3 v_idx = index_buffer[primIdx];

  const float3 v0   = vertex_buffer[ v_idx.x ];
  const float3 v1   = vertex_buffer[ v_idx.y ];
  const float3 v2   = vertex_buffer[ v_idx.z ];
  const float  area = length(cross(v1-v0, v2-v0));

  optix::Aabb* aabb = (optix::Aabb*)result;
  
  if(area > 0.0f && !isinf(area)) {
    aabb->m_min = fminf( fminf( v0, v1), v2 );
    aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
  } else {
    aabb->invalidate();
  }
}

