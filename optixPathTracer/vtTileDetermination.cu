#include "hip/hip_runtime.h"
#include <optix.h>
#include "vtHelpers.h"

using namespace optix;

rtDeclareVariable(uint, launch_index, rtLaunchIndex, );
rtBuffer<uint, 1>  tileID_to_frameID_buffer;
rtBuffer<uint3, 1> unique_tileID_buffer;
rtDeclareVariable(uint, frameID, , );

// This kernel was based on work by Hollemeersch et al. 2010
RT_PROGRAM void list_marked_tiles_kernel()
{
	// If the tile with current index was visible this frame...
    if (tileID_to_frameID_buffer[launch_index] == frameID)
	{
		// Increase the unique tile counter at index 0 with an atomic operation.
        // Note that atomic add returns value found at given memory location BEFORE addition.
		uint unique_tileID_buffer_index = 1u + atomicAdd(&(unique_tileID_buffer[0u].x), 1u);
		size_t unique_tileID_buffer_size = unique_tileID_buffer.size();

		//  Write tileID to unique_tileID_buffer (if any room left).
		if (unique_tileID_buffer_index < make_int1(unique_tileID_buffer_size).x)
		{
			// Translate tile index back to tileID.
			uint mipID, x, y;
            tileIDForOffset(launch_index, mipID, x, y);
			uint3 tileID = make_uint3(mipID, x, y);

			// And store the tileID in the buffer.
			unique_tileID_buffer[unique_tileID_buffer_index] = tileID;
		}
	}
}
