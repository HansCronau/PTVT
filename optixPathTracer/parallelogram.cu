#include "hip/hip_runtime.h"
/*
 * Path Traced Virtual Textures (PTVT)
 * Copyright 2018 Hans Cronau
 *
 * Demo based on the Optix SDK optixPathTracer sample,
 * Copyright 2016 Nvidia Corporation
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float4, plane, , );
// Note: v1 and v2 are defined as offset1 and offset2 (vectors pointing from the anchor to adjacent corners of the parallelogram),
//       and divided by their original length squared (i.e. vector dotted with itself).
//       They are used to create ortogonal vector projections onto respectively offset1 and offset2 (for points defined relative to the anchor).
//       Math of projection vector a onto vector b: a_projected = dot(a,b)/lenth(b)^2*b
//       Division by the length of the vector squared is done because:
//         1. The dot product is divided by the length of b to find the scalar product (= length of the projection vector).
//            b normalised to unit length can be multiplied by this length to find the projection vector.
//            b is normalised by dividing it by its own length. This division is moved to front and combined with the other division by length.
//         2. This combination (the square of the length b) is easily calculated as the dot product of b with itself.
//            There's no need for any (computationally hard) square roots.
//       To empasise: v1 and v2 are NOT the edges of the parallelogram.
rtDeclareVariable(float3, v1, , );
rtDeclareVariable(float3, v2, , );
rtDeclareVariable(float3, anchor, , );
rtDeclareVariable(float3, offset1, , );
rtDeclareVariable(float3, offset2, , );
rtDeclareVariable(int, lgt_instance, , ) = { 0 };

rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(int, lgt_idx, attribute lgt_idx, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

// Path differentials
rtDeclareVariable(float2, T_alpha, attribute T_alpha, );
rtDeclareVariable(float2, T_beta, attribute T_beta, );
rtDeclareVariable(float2, T_gamma, attribute T_gamma, );
rtDeclareVariable(float3, non_normalised_normal, attribute non_normalised_normal, );
rtDeclareVariable(float3, E_0, attribute E_0, );
rtDeclareVariable(float3, E_2, attribute E_2, );

// Debug - Barycentric coordinates
rtDeclareVariable(float3, P_alpha, attribute P_alpha, );
rtDeclareVariable(float3, P_gamma, attribute P_gamma, );

RT_PROGRAM void intersect(int primIdx)
{
    float3 n = make_float3(plane);
    float dt = dot(ray.direction, n);
    float t = (plane.w - dot(n, ray.origin)) / dt;
    if (t > ray.tmin && t < ray.tmax) {
        float3 p = ray.origin + ray.direction * t;
        float3 vi = p - anchor;
        float a1 = dot(v1, vi);
        if (a1 >= 0 && a1 <= 1){
            float a2 = dot(v2, vi);
            if (a2 >= 0 && a2 <= 1){
                if (rtPotentialIntersection(t)) {
					shading_normal = geometric_normal = n;

					// From this point on we'll treat the parallelogram as a (double) triangle.
					// This treatment makes it easier to apply the same differential calculations for both triangles and parallelograms.

					const float3 p0 = anchor;
					const float3 p1 = anchor + offset1;
					const float3 p2 = anchor + offset2;

					const float3 e0 = p1 - p0;
					const float3 e1 = p0 - p2;
                    // Note: e0 and e1 are named by OptiX convention. They correspond to E_0 and E_2 respectively.
					texcoord = make_float3(a1, a2, 0);

                    // Differentials
					// Add triangle-like texture coordinates.
					T_alpha = make_float2(0, 0);
					T_beta  = make_float2(1, 0);
					T_gamma = make_float2(0, 1);

                    non_normalised_normal = cross(e1, e0);
					E_0 = e0; // = p1 - p0
					E_2 = e1; // = p0 - p2

                    // Debug - Barycentric coordinates
                    P_alpha = p0;
                    P_gamma = p2;

                    lgt_idx = lgt_instance;
                    rtReportIntersection(0);
                }
            }
        }
    }
}

RT_PROGRAM void bounds(int, float result[6])
{
    // v1 and v2 are scaled by 1./length^2.  Rescale back to normal for the bounds computation.
    const float3 tv1 = v1 / dot(v1, v1);
    const float3 tv2 = v2 / dot(v2, v2);
    const float3 p00 = anchor;
    const float3 p01 = anchor + tv1;
    const float3 p10 = anchor + tv2;
    const float3 p11 = anchor + tv1 + tv2;
    const float  area = length(cross(tv1, tv2));

    optix::Aabb* aabb = (optix::Aabb*)result;

    if (area > 0.0f && !isinf(area)) {
        aabb->m_min = fminf(fminf(p00, p01), fminf(p10, p11));
        aabb->m_max = fmaxf(fmaxf(p00, p01), fmaxf(p10, p11));
    }
    else {
        aabb->invalidate();
    }
}
